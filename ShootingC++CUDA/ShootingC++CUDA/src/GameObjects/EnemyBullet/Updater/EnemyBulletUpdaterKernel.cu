#include "hip/hip_runtime.h"
﻿#include "EnemyBulletUpdaterKernel.cuh"

#include "../../../Screen/Screen.h"

__global__ void EnemyBulletUpdaterKernel::Process(EnemyBullet* ebPtr, int length)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx >= length) return;
	// 移動
	ebPtr[idx].m_position += ebPtr[idx].m_speed;

	// 画面外に出たら死亡
	if (ebPtr[idx].m_position.x > Screen::Width + ebPtr[idx].m_collisionRadius ||
		ebPtr[idx].m_position.x < -ebPtr[idx].m_collisionRadius ||
		ebPtr[idx].m_position.y > Screen::Height + ebPtr[idx].m_collisionRadius ||
		ebPtr[idx].m_position.y < -ebPtr[idx].m_collisionRadius) ebPtr[idx].m_isDead = true;
	return;
}