﻿#include "EnemyBulletUpdater.h"

#include <hip/hip_runtime.h>

#include "../EnemyBullet.h"
#include "EnemyBulletUpdaterKernel.cuh"

void EnemyBulletUpdater::Run()
{
	int length = EnemyBulletPtr::length;

	// CPUからGPUにデータを転送
	hipMemcpyAsync(
		EnemyBulletPtr::device,
		EnemyBulletPtr::host,
		sizeof(EnemyBullet) * length,
		hipMemcpyHostToDevice,
		EnemyBulletPtr::stream);

	dim3 block(256, 1, 1);
	dim3 grid((length + 256 - 1) / 256, 1, 1);

	// 敵弾更新カーネル実行
	EnemyBulletUpdaterKernel::Process << <grid, block, 0, EnemyBulletPtr::stream >> > (EnemyBulletPtr::device, length);

	// GPUからCPUにデータを転送
	hipMemcpyAsync(
		EnemyBulletPtr::host,
		EnemyBulletPtr::device,
		sizeof(EnemyBullet) * length,
		hipMemcpyDeviceToHost,
		EnemyBulletPtr::stream);
}