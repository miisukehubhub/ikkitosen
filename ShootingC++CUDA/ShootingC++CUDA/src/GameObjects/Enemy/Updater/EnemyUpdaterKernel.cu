#include "hip/hip_runtime.h"
﻿#include "EnemyUpdaterKernel.cuh"

#include <math.h>

#include "../../../Screen/Screen.h"
#include "../EnemyKind.h"
#include "../../Player/Player.h"

__global__ void EnemyUpdaterKernel::Process(Enemy* ePtr, int length)
{
	// スレッド番号の取得
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	// 存在しない要素は処理を行わない
	if (idx >= length) return;

	ePtr[idx].m_state &= ~(unsigned char)EnemyState::IsDamage;
	ePtr[idx].m_state &= ~(unsigned char)EnemyState::canFire;

	if (ePtr[idx].m_kind == EnemyKind::SoftEnemy) SoftEnemyUpdate(&ePtr[idx]);
	if (ePtr[idx].m_kind == EnemyKind::HardEnemy) HardEnemyUpdate(&ePtr[idx]);
	if (ePtr[idx].m_kind == EnemyKind::MetalEnemy) MetalEnemyUpdate(&ePtr[idx]);

	if (ePtr[idx].m_position.x < -ePtr[idx].m_collisionRadius)
	{
		// 画面外に出ての死亡はスコアが入らないようにする
		ePtr[idx].m_state |= (unsigned char)EnemyState::IsDead;
		ePtr[idx].m_score = 0;
	}
	return;
}

__device__ void EnemyUpdaterKernel::SoftEnemyUpdate(Enemy* ePtr)
{
	ePtr->m_angle += 0.05f;

	ePtr->m_basePosition += ePtr->m_speed;
	ePtr->m_position = ePtr->m_basePosition;
	ePtr->m_position.y += sinf(ePtr->m_angle) * 30.0f;

	if (ePtr->m_bulletFireIntervalTimer > 0)
	{
		ePtr->m_bulletFireIntervalTimer--;
		return;
	}

	ePtr->m_state |= (unsigned char)EnemyState::canFire;
	ePtr->m_bulletFireIntervalTimer = ePtr->m_bulletFireIntervalTimeFrame;
}

__device__ void EnemyUpdaterKernel::HardEnemyUpdate(Enemy* ePtr)
{
	ePtr->m_angle += 0.05f;

	ePtr->m_basePosition += ePtr->m_speed;
	ePtr->m_position = ePtr->m_basePosition;
	ePtr->m_position.x += cosf(ePtr->m_angle) * 20.0f;

	if (ePtr->m_bulletFireIntervalTimer > 0)
	{
		ePtr->m_bulletFireIntervalTimer--;
		return;
	}

	ePtr->m_state |= (unsigned char)EnemyState::canFire;
	ePtr->m_bulletFireIntervalTimer = ePtr->m_bulletFireIntervalTimeFrame;
}

__device__ void EnemyUpdaterKernel::MetalEnemyUpdate(Enemy* ePtr)
{
	ePtr->m_angle += 0.03f;

	ePtr->m_basePosition += ePtr->m_speed;
	ePtr->m_position = ePtr->m_basePosition;
	ePtr->m_position.x += cosf(ePtr->m_angle) * 32.0f;
	ePtr->m_position.y += sinf(ePtr->m_angle) * 32.0f;

	if (ePtr->m_bulletFireIntervalTimer > 0)
	{
		ePtr->m_bulletFireIntervalTimer--;
		return;
	}

	ePtr->m_state |= (unsigned char)EnemyState::canFire;
	ePtr->m_bulletFireIntervalTimer = ePtr->m_bulletFireIntervalTimeFrame;
}