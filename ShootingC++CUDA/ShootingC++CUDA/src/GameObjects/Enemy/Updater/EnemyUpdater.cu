﻿#include "EnemyUpdater.h"

#include <hip/hip_runtime.h>

#include "../Enemy.h"
#include "EnemyUpdaterKernel.cuh"

void EnemyUpdater::Run()
{
	int length = EnemyPtr::length;

	// CPUからGPUにデータを転送
	hipMemcpyAsync(
		EnemyPtr::device,
		EnemyPtr::host,
		sizeof(Enemy) * length,
		hipMemcpyHostToDevice,
		EnemyPtr::stream);

	dim3 block(256, 1, 1);
	dim3 grid((length + 256 - 1) / 256, 1, 1);

	// 敵の更新カーネルを実行
	EnemyUpdaterKernel::Process << <grid, block, 0, EnemyPtr::stream >> > (EnemyPtr::device, length);

	// GPUからCPUにデータを転送
	hipMemcpyAsync(
		EnemyPtr::host,
		EnemyPtr::device,
		sizeof(Enemy) * length,
		hipMemcpyDeviceToHost,
		EnemyPtr::stream);
}