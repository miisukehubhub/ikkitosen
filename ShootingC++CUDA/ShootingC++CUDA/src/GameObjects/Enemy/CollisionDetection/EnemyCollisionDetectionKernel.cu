#include "hip/hip_runtime.h"
﻿#include "EnemyCollisionDetectionKernel.cuh"

#include "../../../MyMath/MyMath.h"

#include "../../PlayerBullets/PlayerNormalBullet/PlayerNormalBullet.h"
#include "../../PlayerBullets/PlayerScatterBullet/PlayerScatterBullet.h"

__global__ void EnemyCollisionDetectionKernel::Process(Enemy* ePtr, int eLength, PlayerNormalBullet* pnbPtr, int pnbLength, PlayerScatterBullet* psbPtr, int psbLength)
{
	 // スレッド番号を取得
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	// 存在しない要素は処理を行わない
	if (idx >= eLength) return;
	
	for (int i = 0; i < pnbLength; i++)
	{
		// 敵が死んでいたら終了
		if ((ePtr[idx].m_state & (unsigned char)EnemyState::IsDead) != 0) return;

		if (MyMath::CircleCircleIntersection(ePtr[idx].m_position, ePtr[idx].m_collisionRadius,
											  pnbPtr[i].m_position, pnbPtr[i].m_collisionRadius))
		{
			int damage = pnbPtr[i].m_power / (ePtr[idx].m_defenceLevel + 1);
			ePtr[idx].m_hp -= damage;
			if (damage > 0) ePtr[idx].m_state |= (unsigned char)EnemyState::IsDamage;

			if (ePtr[idx].m_hp < 0)
			{
				ePtr[idx].m_state |= (unsigned char)EnemyState::IsDead;
			}
		}
	}

	for (int i = 0; i < psbLength; i++)
	{
		// 敵が死んでいたら終了
		if ((ePtr[idx].m_state & (unsigned char)EnemyState::IsDead) != 0) return;

		if (MyMath::CircleCircleIntersection(ePtr[idx].m_position, ePtr[idx].m_collisionRadius,
			psbPtr[i].m_position, psbPtr[i].m_collisionRadius))
		{
			int damage = pnbPtr[i].m_power / (ePtr[idx].m_defenceLevel + 1);
			ePtr[idx].m_hp -= damage;
			if (damage > 0) ePtr[idx].m_state |= (unsigned char)EnemyState::IsDamage;
			psbPtr[i].m_isDead = true;

			if (ePtr[idx].m_hp < 0)
			{
				ePtr[idx].m_state |= (unsigned char)EnemyState::IsDead;
			}
		}
	}

	return;
}