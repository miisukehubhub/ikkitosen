﻿#include "EnemyCollisionDetection.h"

#include <hip/hip_runtime.h>

#include "../Enemy.h"
#include "EnemyCollisionDetectionKernel.cuh"
#include "../../PlayerBullets/PlayerNormalBullet/PlayerNormalBullet.h"
#include "../../PlayerBullets/PlayerScatterBullet/PlayerScatterBullet.h"

void EnemyCollisionDetection::Run()
{
	int eLength = EnemyPtr::length;
	int pnbLength = PlayerNormalBulletPtr::length;
	int psbLength = PlayerScatterBulletPtr::length;

	dim3 block(256, 1, 1);
	dim3 grid((eLength + 256 - 1) / 256 * 4, 1, 1);

	// 敵の当たり判定カーネルを実行
	EnemyCollisionDetectionKernel::Process << <grid, block, 0, EnemyPtr::stream >> > (EnemyPtr::device, eLength, PlayerNormalBulletPtr::device, pnbLength, PlayerScatterBulletPtr::device, psbLength);

	// GPUからCPUにデータを転送
	hipMemcpyAsync(
		EnemyPtr::host,
		EnemyPtr::device,
		sizeof(Enemy) * eLength,
		hipMemcpyDeviceToHost,
		EnemyPtr::stream);

	hipMemcpyAsync(
		PlayerNormalBulletPtr::host,
		PlayerNormalBulletPtr::device,
		sizeof(PlayerNormalBullet) * pnbLength,
		hipMemcpyDeviceToHost,
		EnemyPtr::stream);

	hipMemcpyAsync(
		PlayerScatterBulletPtr::host,
		PlayerScatterBulletPtr::device,
		sizeof(PlayerScatterBullet) * psbLength,
		hipMemcpyDeviceToHost,
		EnemyPtr::stream);
}