﻿#include "PlayerScatterBulletUpdater.h"

#include <hip/hip_runtime.h>

#include "../PlayerScatterBullet.h"
#include "PlayerScatterBulletUpdaterKernel.cuh"

void PlayerScatterBulletUpdater::Run()
{
	int length = PlayerScatterBulletPtr::length;

	// CPUからGPUにデータを転送
	hipMemcpyAsync(
		PlayerScatterBulletPtr::device,
		PlayerScatterBulletPtr::host,
		sizeof(PlayerScatterBullet) * length,
		hipMemcpyHostToDevice,
		PlayerScatterBulletPtr::stream);

	dim3 block(256, 1, 1);
	dim3 grid((length + 256 - 1) / 256, 1, 1);

	// プレイヤー散弾更新カーネル実行
	PlayerScatterBulletUpdaterKernel::Process << <grid, block, 0, PlayerScatterBulletPtr::stream >> > (PlayerScatterBulletPtr::device, length);

	// GPUからCPUにデータを転送
	hipMemcpyAsync(
		PlayerScatterBulletPtr::host,
		PlayerScatterBulletPtr::device,
		sizeof(PlayerScatterBullet) * length,
		hipMemcpyDeviceToHost,
		PlayerScatterBulletPtr::stream);
}