#include "hip/hip_runtime.h"
﻿#include "PlayerScatterBulletUpdaterKernel.cuh"

#include "../../../../Screen/Screen.h"

__global__ void PlayerScatterBulletUpdaterKernel::Process(PlayerScatterBullet* psbPtr, int length)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx >= length) return;

	psbPtr[idx].m_position += psbPtr[idx].m_speed; // 移動

	if (psbPtr[idx].m_position.x > Screen::Width + psbPtr[idx].m_visibleRadius ||
		psbPtr[idx].m_position.y < 0 - psbPtr[idx].m_visibleRadius ||
		psbPtr[idx].m_position.y > Screen::Height + psbPtr[idx].m_visibleRadius)
	{
		psbPtr[idx].m_isDead = true;
	}
	return;
}