#include "hip/hip_runtime.h"
﻿#include "PlayerNormalBulletUpdaterKernel.cuh"

#include "../../../../Screen/Screen.h"

__global__ void PlayerNormalBulletUpdaterKernel::Process(PlayerNormalBullet* pnbPtr, int length)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx >= length) return;

	pnbPtr[idx].m_position += pnbPtr[idx].m_speed; // 移動

	// 画面の右端を超えたら死亡
	if (pnbPtr[idx].m_position.x > Screen::Width + pnbPtr[idx].m_visibleRadius) pnbPtr[idx].m_isDead = true;
	return;
}