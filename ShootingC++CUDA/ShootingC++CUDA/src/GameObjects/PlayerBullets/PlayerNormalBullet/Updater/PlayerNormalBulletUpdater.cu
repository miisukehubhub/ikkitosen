﻿#include "PlayerNormalBulletUpdater.h"

#include <hip/hip_runtime.h>

#include "../PlayerNormalBullet.h"
#include "PlayerNormalBulletUpdaterKernel.cuh"

void PlayerNormalBulletUpdater::Run()
{
	int length = PlayerNormalBulletPtr::length;

	// CPUからGPUにデータを転送
	hipMemcpyAsync(
		PlayerNormalBulletPtr::device,
		PlayerNormalBulletPtr::host,
		sizeof(PlayerNormalBullet) * length,
		hipMemcpyHostToDevice,
		PlayerNormalBulletPtr::stream);

	dim3 block(256, 1, 1);
	dim3 grid((length + 256 - 1) / 256, 1, 1);

	// プレイヤー通常弾更新カーネルを実行
	PlayerNormalBulletUpdaterKernel::Process << <grid, block, 0, PlayerNormalBulletPtr::stream >> > (PlayerNormalBulletPtr::device, length);

	// GPUからCPUにデータを転送
	hipMemcpyAsync(
		PlayerNormalBulletPtr::host,
		PlayerNormalBulletPtr::device,
		sizeof(PlayerNormalBullet) * length,
		hipMemcpyDeviceToHost,
		PlayerNormalBulletPtr::stream);
}